#include "hip/hip_runtime.h"
#include "SparseSolver_GPU.cuh"

void solve_ls_cg_gpu(int size, int constraints, double* A, int* columns, int* rowindex, double* x, double* b, int nonzeros, double tolerance, int maxitrs)
{
	/* Create CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);

	checkCudaErrors(hipblasStatus_t);

	/* Create CUSPARSE context */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	cusparseStatus = hipsparseCreate(&cusparseHandle);

	checkCudaErrors(cusparseStatus);

	/* Description of the A matrix*/
	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);

	checkCudaErrors(cusparseStatus);

	/* Define the properties of the matrix */
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ONE);

	int* dACol;
	int* dARow;
	int* dATACol;
	int* dATARow;
	double* dA;
	double* dATA;
	double* db;
	double* dATb;
	int nnzA = nonzeros;
	int nA = size;
	int ncA = constraints;
	int nnzATA = 0;
	int nATA = size;

	/* Allocate required memory */
	checkCudaErrors(hipMalloc((void **)&dACol,  nnzA * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&dARow,  (ncA + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&dA,     nnzA * sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&db,     ncA * sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&dATb,   nA * sizeof(double)));

	hipMemcpy(dACol, columns,   nnzA * sizeof(int),      hipMemcpyHostToDevice);
	hipMemcpy(dARow, rowindex,  (ncA + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dA,    A,         nnzA * sizeof(double),   hipMemcpyHostToDevice);
	hipMemcpy(db,    b,         ncA * sizeof(double),    hipMemcpyHostToDevice);

	/* Compute ATA */
	hipsparseMatDescr_t aDescr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&aDescr);

	checkCudaErrors(cusparseStatus);

	/* Define the properties of the matrix */
	hipsparseSetMatType(aDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(aDescr, HIPSPARSE_INDEX_BASE_ONE);

	int *nnzTotalDevHostPtr = &nnzATA;
	hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST);
	hipMalloc((void**)&dATARow, sizeof(int) * (nATA + 1));

	hipsparseXcsrgemmNnz(cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nA, nA, ncA,
		aDescr, nnzA, dARow, dACol, aDescr, nnzA, dARow, dACol, descr, dATARow, nnzTotalDevHostPtr);

	if (NULL != nnzTotalDevHostPtr)
	{ 
		nnzATA = *nnzTotalDevHostPtr;
	}
	else
	{
		hipMemcpy(&nnzATA, dATARow + nA, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&nATA,   dATARow + nA, sizeof(int), hipMemcpyDeviceToHost);
		nnzATA -= nATA;
	}

	hipMalloc((void**)&dATACol, sizeof(int)* nnzATA);
	hipMalloc((void**)&dATA,    sizeof(double)* nnzATA);

	hipsparseDcsrgemm(cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, nA, nA, ncA,
		aDescr, nnzA, dA, dARow, dACol, aDescr, nnzA, dA, dARow, dACol, descr, dATA, dATARow, dATACol);

	hipsparseDestroyMatDescr(aDescr);
	
	/* Compute ATb */
	double alpha = 1.0;
	double beta = 0.0;

	cusparseStatus = hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, ncA, nA, nnzA, &alpha, descr, dA, dARow, dACol, db, &beta, dATb);

	/* Release useless memory */
	hipFree(dA);
	hipFree(db);
	hipFree(dACol);
	hipFree(dARow);

	double* dx;
	double* dP;
	double* dAx;

	/* Allocate required memory */
	checkCudaErrors(hipMalloc((void **)&dx, nATA * sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&dP, nATA * sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&dAx, nATA * sizeof(double)));

	hipMemcpy(dx, x, nATA * sizeof(double), hipMemcpyHostToDevice);

	/* Conjugate gradient without preconditioning.
	------------------------------------------
	Follows the description by Golub & Van Loan, "Matrix Computations 3rd ed.", Section 10.2.6  */
	printf("Convergence of conjugate gradient without preconditioning: \n");

	int k = 0;
	double r0 = 0, r1 = 0, dot = 0;
	double alpham1 = -1.0;
	double vb = 0.0;
	double va = 0.0, na = 0.0;

	alpha = 1.0;
	beta = 0.0;

	hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nATA, nATA, nnzATA, &alpha, descr, dATA, dATARow, dATACol, dx, &beta, dAx);
	hipblasDaxpy(cublasHandle, nATA, &alpham1, dAx, 1, dATb, 1);
	hipblasStatus_t = hipblasDdot(cublasHandle, nATA, dATb, 1, dATb, 1, &r1);

	k = 1;
	while (r1 > tolerance * tolerance && k <= maxitrs)
	{
		if (k > 1)
		{
			vb = r1 / r0;
			hipblasStatus_t = hipblasDscal(cublasHandle, nATA, &vb, dP, 1);
			hipblasStatus_t = hipblasDaxpy(cublasHandle, nATA, &alpha, dATb, 1, dP, 1);
		}
		else
		{
			hipblasStatus_t = hipblasDcopy(cublasHandle, nATA, dATb, 1, dP, 1);
		}

		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nATA, nATA, nnzATA, &alpha, descr, dATA, dATARow, dATACol, dP, &beta, dAx);
		hipblasDdot(cublasHandle, nATA, dP, 1, dAx, 1, &dot);

		va = r1 / dot;
		hipblasDaxpy(cublasHandle, nATA, &va, dP, 1, dx, 1);

		na = -va;
		hipblasDaxpy(cublasHandle, nATA, &na, dAx, 1, dATb, 1);

		r0 = r1;
		hipblasDdot(cublasHandle, nATA, dATb, 1, dATb, 1, &r1);
		hipDeviceSynchronize();
		k++;
	}

	//printf("  iteration = %3d, residual = %e \n", k, sqrt(r1));

	hipMemcpy(x, dx, nATA * sizeof(double), hipMemcpyDeviceToHost);

	/* check result */
	if (0)
	{
		double err = 0.0;
		for (int i = 0; i < nATA; i++)
		{
			double rsum = 0.0;
			for (int j = rowindex[i]; j < rowindex[i + 1]; j++)
			{
				rsum += A[j - 1] * x[columns[j - 1] - 1];
			}

			double diff = fabs(rsum - b[i]);

			if (diff > err)
			{
				err = diff;
			}
		}
		printf("  Convergence Test: %s \n", (k <= maxitrs) ? "OK" : "FAIL");
	}

	hipsparseDestroy(cusparseHandle);
	hipblasDestroy(cublasHandle);
	hipsparseDestroyMatDescr(descr);

	hipFree(dATACol);
	hipFree(dATARow);
	hipFree(dATA);
	hipFree(dx);
	hipFree(dATb);
	hipFree(dP);
	hipFree(dAx);
}